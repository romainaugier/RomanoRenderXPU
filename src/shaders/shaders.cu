#include "hip/hip_runtime.h"
#include "romanorender/optix_params.h"
#include "romanorender/ray.h"

#include "payload.cuh"
#include "random.cuh"
#include "mat44.cuh"
#include "float4.cuh"
#include "float3.cuh"
#include "sampling.cuh"

#include <hiprand/hiprand_kernel.h>
#include <optix.h>
#include <hip/hip_vector_types.h>

extern "C" __constant__ OptixParams params;

__device__ float2 get_pmj02_sample(const uint pixel_idx, const uint sample) noexcept
{
    return params.pmj_samples[pixel_idx % NUM_PMJ02_SEQUENCES][sample];
}

__device__ float3 get_primary_ray_dir(const float aspect, 
                                      const float fov,
                                      const Mat44F& transform, 
                                      const float rx, 
                                      const float ry)
{
    const uint3 launch_index = optixGetLaunchIndex();
    const uint3 launch_dims = optixGetLaunchDimensions();

    const float ndc_x = (2.0f * ((float)launch_index.x + rx) / (float)launch_dims.x - 1.0f) * aspect;
    const float ndc_y = 1.0f - 2.0f * ((float)launch_index.y + ry) / (float)launch_dims.y;

    const float tan_half_fov = __tanf(deg2radf(fov * 0.5f));
    const float px = ndc_x * tan_half_fov;
    const float py = ndc_y * tan_half_fov;

    const float3 direction = make_float3(px, py, -1.0f);

    return normalize_float3(transform.transform_dir(direction));
}

extern "C" __global__ void __raygen__rg()
{
    const uint3 launch_index = optixGetLaunchIndex();
    const uint3 launch_dims = optixGetLaunchDimensions();

    const uint pixel_idx = launch_index.x + launch_index.y * launch_dims.x;
    const uint sample_idx = params.current_sample + launch_index.z;

    RayData ray_data;

    const Mat44F transform(params.camera_transform);

    const float2 random_sample = get_pmj02_sample(pixel_idx, sample_idx);
    const float2 filter = sample_gaussian(random_sample.x, random_sample.y);
    const float3 ray_dir = get_primary_ray_dir(params.camera_aspect, params.camera_fov, transform, random_sample.x, random_sample.y);
    const float3 ray_pos = make_float3(params.camera_transform[12], params.camera_transform[13], params.camera_transform[14]);

    uint2 payload = split_ptr(&ray_data);

    optixTrace(params.handle,
               ray_pos,
               ray_dir,
               0.0f,
               1e16f,
               0.0f,
               OptixVisibilityMask(romanorender::VisibilityFlag_VisiblePrimaryRays),
               OPTIX_RAY_FLAG_NONE,
               0,
               1,
               0,
               payload.x,
               payload.y);

    params.pixels[pixel_idx] = lerp_float4f(params.pixels[pixel_idx],
                                            ray_data.color,
                                            1.0f / (float)(params.current_sample + launch_index.z));
}

extern "C" __global__ void __miss__ms()
{
    RayData* ray_data = merge_ptr(optixGetPayload_0(), optixGetPayload_1());

    ray_data->color = make_float4(0.0f);
}

__device__ float3 get_normal(const GeometryData* geom_data, 
                             const unsigned int primitive,
                             const float2 uv)
{
    const uint3 indices = geom_data->indices[primitive];

    if(geom_data->normals == 0)
    {
        const float4 v0 = geom_data->vertices[indices.x];
        const float4 v1 = geom_data->vertices[indices.y];
        const float4 v2 = geom_data->vertices[indices.z];

        const float4 edge0 = v1 - v0;
        const float4 edge1 = v2 - v0;
        const float4 object_normal = normalize_float4(cross_float4(edge0, edge1));

        return make_float3(object_normal);
    }
    else
    {
        const float3 n0 = geom_data->normals[indices.x];
        const float3 n1 = geom_data->normals[indices.y];
        const float3 n2 = geom_data->normals[indices.z];

        const float w = 1.0f - uv.x - uv.y;

        return n0 * w + n1 * uv.x + n2 * uv.y;
    }
}

extern "C" __global__ void __closesthit__ch()
{
    RayData* ray_data = merge_ptr(optixGetPayload_0(), optixGetPayload_1());

    const GeometryData* geom_data = reinterpret_cast<const GeometryData*>(optixGetSbtDataPointer());

    const float3 normal = optixTransformNormalFromObjectToWorldSpace(get_normal(geom_data, 
                                                                                optixGetPrimitiveIndex(),
                                                                                optixGetTriangleBarycentrics()));

    const float3 color = (normalize_float3(normal) + 0.5f) / 2.0f;

    ray_data->color = make_float4(color, 1.0f);
}