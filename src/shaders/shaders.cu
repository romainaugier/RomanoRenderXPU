#include "hip/hip_runtime.h"
#include "romanorender/optix_params.h"

#include "float3.cuh"
#include "float4.cuh"
#include "payload.cuh"
#include "random.cuh"

#include <hiprand/hiprand_kernel.h>
#include <optix.h>
#include <hip/hip_vector_types.h>

extern "C" __constant__ OptixParams params;

__device__ float3 mat44f_mul_dir(const float* M, const float3& v) noexcept
{
    return make_float3(v.x * M[0] + v.y * M[4] + v.z * M[8],
                       v.x * M[1] + v.y * M[5] + v.z * M[9],
                       v.x * M[2] + v.y * M[6] + v.z * M[10]);
}

__device__ float3 get_ray_dir(float aspect, float fov, float* transform, float rx, float ry)
{
    uint3 launch_index = optixGetLaunchIndex();
    uint3 launch_dims = optixGetLaunchDimensions();

    const float ndc_x = (2.0f * ((float)launch_index.x + rx) / (float)launch_dims.x - 1.0f) * aspect;
    const float ndc_y = 1.0f - 2.0f * ((float)launch_index.y + ry) / (float)launch_dims.y;

    const float tan_half_fov = __tanf(deg2radf(fov * 0.5f));
    const float px = ndc_x * tan_half_fov;
    const float py = ndc_y * tan_half_fov;

    float3 direction = make_float3(px, py, -1.0f);

    return normalize_float3(mat44f_mul_dir(transform, direction));
}

extern "C" __global__ void __raygen__rg()
{
    uint3 launch_index = optixGetLaunchIndex();
    uint3 launch_dims = optixGetLaunchDimensions();

    unsigned long long seed = params.seed;
    unsigned long long sequence = launch_index.x + launch_index.y * launch_dims.x;
    unsigned long long offset = params.current_sample;

    float rand_x = random_float_01(seed + sequence + offset);
    float rand_y = random_float_01(seed + sequence + offset + 1);

    RayData ray_data;

    float3 ray_dir = get_ray_dir(params.camera_aspect, params.camera_fov, params.camera_transform, rand_x, rand_y);
    float3 ray_pos = make_float3(params.camera_transform[3], params.camera_transform[7], params.camera_transform[11]);

    uint2 payload = split_ptr(&ray_data);

    optixTrace(params.handle, // Scene acceleration structure
               ray_pos,       // Ray origin
               ray_dir,       // Ray direction
               0.0f,          // tMin
               1e16f,         // tMax
               0.0f,          // Ray time (for motion blur, unused here)
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_NONE,
               0,
               1,
               0,
               payload.x,
               payload.y);

    unsigned int pixel_idx = launch_index.x + launch_index.y * launch_dims.x;
    params.pixels[pixel_idx]
        = lerp_float4f(params.pixels[pixel_idx], ray_data.color, 1.0f / (float)params.current_sample);
}

extern "C" __global__ void __miss__ms()
{
    RayData* ray_data = merge_ptr(optixGetPayload_0(), optixGetPayload_1());

    ray_data->color = make_float4(0.0f);
}

extern "C" __global__ void __closesthit__ch()
{
    RayData* ray_data = merge_ptr(optixGetPayload_0(), optixGetPayload_1());
    const GeometryData* geomData = reinterpret_cast<const GeometryData*>(optixGetSbtDataPointer());

    const uint3 indices = geomData->indices[optixGetPrimitiveIndex()];

    const float4 v0 = geomData->vertices[indices.x];
    const float4 v1 = geomData->vertices[indices.y];
    const float4 v2 = geomData->vertices[indices.z];

    const float4 edge0 = v1 - v0;
    const float4 edge1 = v2 - v0;
    const float4 objectNormal = normalize_float4(cross_float4(edge0, edge1));

    const float3 normal
        = optixTransformNormalFromObjectToWorldSpace(make_float3(objectNormal.x, objectNormal.y, objectNormal.z));

    const float3 color = (normal + 0.5f) / 2.0f;

    ray_data->color = make_float4(color, 1.0f);
}